#include "hip/hip_runtime.h"
#include <iostream>
#include <Random123/philox.h>
#include <Random123/ReinterpretCtr.hpp>

#include "example_seeds.h"
#include "util_cuda.h"

using namespace r123;
using namespace std;

typedef long unsigned uint64_t;
#define THREADS_PER_BLOCK 32

template<typename T>
__global__ void
uniform_ct_gpu(unsigned useed,
               T* arr) {

    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    typedef Philox4x64 G;
    G rng;
    G::key_type k = {{tid, useed}};
    G::ctr_type c = {{}};

    union {
        G::ctr_type c;
        int4 i;
    }u;
    c.incr();
    u.c = rng(c, k);

    if (std::is_same<T, int64_t>::value){
        arr[4*tid]   = u.i.x;
        arr[4*tid+1] = u.i.y;
        arr[4*tid+2] = u.i.z;
        arr[4*tid+3] = u.i.w;
    }else{
        arr[4*tid]   = ((double)((uint64_t)u.i.x))/((double)UINT_MAX);
        arr[4*tid+1] = ((double)((uint64_t)u.i.y))/((double)UINT_MAX);
        arr[4*tid+2] = ((double)((uint64_t)u.i.z))/((double)UINT_MAX);
        arr[4*tid+3] = ((double)((uint64_t)u.i.w))/((double)UINT_MAX);
    }
}

extern unsigned getUseed();

template<typename T>
T *
__generateRandomsGPU_onD(unsigned long N) {
    assert(N%4 ==0);
    unsigned useed = getUseed();
    T * randomNumbers_d;

    size_t rn_size = N * sizeof(T);

    CHECKCALL(hipMalloc(& randomNumbers_d, rn_size));

    unsigned threads_per_block = THREADS_PER_BLOCK;
    assert(N%THREADS_PER_BLOCK == 0);
    unsigned blocks_per_grid   = N / threads_per_block;
    cerr << threads_per_block << endl;
    cerr << blocks_per_grid   << endl;
    cerr << N                 << endl;

    if (std::is_same<T, int64_t>::value)
        cerr << "VALUE is int64_t "<<endl;
    else
        cerr << "VALUE is double"<<endl;

    uniform_ct_gpu<<<blocks_per_grid, threads_per_block>>>(
        useed, randomNumbers_d);

    return randomNumbers_d;
}

template<typename T>
T *
__generateRandomsGPU(unsigned long N) {

    T * randomNumbers_d, * randomNumbers_h;

    randomNumbers_h = (T *) malloc(N*sizeof(T));

    randomNumbers_d = __generateRandomsGPU_onD<T>(N);
    CHECKCALL(hipMemcpy(randomNumbers_h, randomNumbers_d,
                N * sizeof(T),
                hipMemcpyDeviceToHost));

    CHECKCALL(hipFree(randomNumbers_d));
    return randomNumbers_h;
}


void deleteRandomsGPU(double * arr){
    CHECKCALL(hipFree(arr));
}

double *
generateRandomsGPUd(unsigned long N){
    return __generateRandomsGPU<double>(N);
}

int64_t *
generateRandomsGPUi(unsigned long N){
    return __generateRandomsGPU<int64_t>(N);
}


#include "Random.h"


class ConusUniformGPU : public galsim::BaseDeviate {

    public:

        ConusUniformGPU(long lseed, int N):
            galsim::BaseDeviate(lseed), buf_len(N), buf_ptr(N) {};
// NOTE: initialize buf_ptr to N so that we're calling fill_buff on the first
// time generate1() is called


        double generate1() {
            buf_ptr++;
            if (buf_ptr < buf_len) return buf_d[buf_ptr];

            hipFree(buf_d);
            fill_buff();

            // Neet to try again after buffer has been filled. This definitely
            // looks unsafe on device. TODO: fix
            return generate1();
        };

    private:
        int buf_len;
        int buf_ptr;

        // NOTE: random numbers are buffered on device!
        double * buf_d;

        void fill_buff(){
            buf_d = __generateRandomsGPU_onD<double>(buf_len);
        };
};
